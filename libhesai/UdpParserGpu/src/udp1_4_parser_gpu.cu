#include "hip/hip_runtime.h"
/************************************************************************************************
Copyright (C) 2023 Hesai Technology Co., Ltd.
Copyright (C) 2023 Original Authors
All rights reserved.

All code in this repository is released under the terms of the following Modified BSD License. 
Redistribution and use in source and binary forms, with or without modification, are permitted 
provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and 
  the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
  the following disclaimer in the documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its contributors may be used to endorse or 
  promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR 
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF 
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
************************************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

#include "udp1_4_parser_gpu.h"
#include "safe_call.cuh"
#include "return_code.h"

using namespace hesai::lidar;
template <typename T_Point>
Udp1_4ParserGpu<T_Point>::Udp1_4ParserGpu() {
  corrections_loaded_ = false;
  cudaSafeMalloc(point_data_cu_, POINT_DATA_LEN);
  cudaSafeMalloc(sensor_timestamp_cu_, SENSOR_TIMESTAMP_LEN);
}
template <typename T_Point>
Udp1_4ParserGpu<T_Point>::~Udp1_4ParserGpu() {
  cudaSafeFree(point_data_cu_);
  cudaSafeFree(sensor_timestamp_cu_);
  if (corrections_loaded_) {
    cudaSafeFree(channel_elevations_cu_);
    cudaSafeFree(channel_azimuths_cu_);
    corrections_loaded_ = false;
  }
}
template <typename T_Point>
__global__ void compute_xyzs_1_4_impl(T_Point *xyzs, const float* channel_azimuths, const float* channel_elevations, const PointDecodeData* point_data, const uint64_t* sensor_timestamp, 
    const double raw_distance_unit, Transform transform, LidarOpticalCenter optical_center, const uint16_t blocknum, const uint16_t lasernum, const uint16_t packet_index, const int fov_start, const int fov_end) {
  float fov_start_rad = fov_start * M_PI / 180.0f;
  float fov_end_rad = fov_end * M_PI / 180.0f;
  auto iscan = blockIdx.x;
  auto ichannel = threadIdx.x;
  if (iscan >= packet_index || ichannel >= blocknum * lasernum) return;
  int point_index = iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum));
  float azimuth = point_data[point_index].azimuth / kHalfCircleFloat * M_PI;
  float theta = (channel_azimuths[(ichannel % lasernum)] * kResolutionFloat) / kHalfCircleFloat * M_PI;
  float phi = (channel_elevations[(ichannel % lasernum)] * kResolutionFloat) / kHalfCircleFloat * M_PI;
  float rho = point_data[point_index].distances * raw_distance_unit;

  if(rho > 0.09 && optical_center.flag) {
    float tx = std::cos(phi) * std::sin(theta);
    float ty = std::cos(phi) * std::cos(theta);
    float tz = std::sin(phi);
    float B = 2 * tx * optical_center.x + 2 * ty * optical_center.y + 2 * tz * optical_center.z;
    float C = optical_center.x * optical_center.x + optical_center.y * optical_center.y + optical_center.z * optical_center.z - rho * rho;
    float d_opitcal = std::sqrt(B * B / 4 - C) - B / 2;
    float x = d_opitcal * tx + optical_center.x;
    float y = d_opitcal * ty + optical_center.y;
    float z = d_opitcal * tz + optical_center.z;
    theta = azimuth + std::atan(x / y);
    phi = std::asin(z / rho);
  } else {
    theta += azimuth;
  }

  if ((fov_start != -1 && theta < fov_start_rad) || (fov_end != -1 && theta > fov_end_rad)) {
    gpu::setX(xyzs[point_index], 0);
    gpu::setY(xyzs[point_index], 0);
    gpu::setZ(xyzs[point_index], 0);
    gpu::setIntensity(xyzs[point_index], 0);
    gpu::setTimestamp(xyzs[point_index], double(sensor_timestamp[iscan]) / kMicrosecondToSecond);
    gpu::setRing(xyzs[point_index], ichannel % lasernum);
    gpu::setConfidence(xyzs[point_index], point_data[point_index].confidence);
    gpu::setWeightFactor(xyzs[point_index], point_data[point_index].weight_factor);
    gpu::setEnvLight(xyzs[point_index], point_data[point_index].env_light);
    return;
  }

  float z = rho * sin(phi);
  auto r = rho * cosf(phi);
  float x = r * sin(theta);
  float y = r * cos(theta);

  float cosa = std::cos(transform.roll);
  float sina = std::sin(transform.roll);
  float cosb = std::cos(transform.pitch);
  float sinb = std::sin(transform.pitch);
  float cosc = std::cos(transform.yaw);
  float sinc = std::sin(transform.yaw);

  float x_ = cosb * cosc * x + (sina * sinb * cosc - cosa * sinc) * y +
              (sina * sinc + cosa * sinb * cosc) * z + transform.x;
  float y_ = cosb * sinc * x + (cosa * cosc + sina * sinb * sinc) * y +
              (cosa * sinb * sinc - sina * cosc) * z + transform.y;
  float z_ = -sinb * x + sina * cosb * y + cosa * cosb * z + transform.z;
  gpu::setX(xyzs[point_index], x_);
  gpu::setY(xyzs[point_index],  y_);
  gpu::setZ(xyzs[point_index], z_);
  gpu::setIntensity(xyzs[point_index], point_data[point_index].reflectivities);
  gpu::setTimestamp(xyzs[point_index], double(sensor_timestamp[iscan]) / kMicrosecondToSecond);
  gpu::setRing(xyzs[point_index], ichannel % lasernum);
  gpu::setConfidence(xyzs[point_index], point_data[point_index].confidence);
  gpu::setWeightFactor(xyzs[point_index], point_data[point_index].weight_factor);
  gpu::setEnvLight(xyzs[point_index], point_data[point_index].env_light);
}
template <typename T_Point>
int Udp1_4ParserGpu<T_Point>::ComputeXYZI(LidarDecodedFrame<T_Point> &frame) {
  if (!corrections_loaded_) return int(ReturnCode::CorrectionsUnloaded);          
  cudaSafeCall(hipMemcpy(point_data_cu_, frame.pointData,
                          frame.block_num * frame.laser_num * frame.packet_num * sizeof(PointDecodeData), 
                          hipMemcpyHostToDevice), ReturnCode::CudaMemcpyHostToDeviceError);
  cudaSafeCall(hipMemcpy(sensor_timestamp_cu_, frame.sensor_timestamp,
                          frame.packet_num * sizeof(uint64_t), 
                          hipMemcpyHostToDevice), ReturnCode::CudaMemcpyHostToDeviceError);   
  compute_xyzs_1_4_impl<<<frame.packet_num, frame.block_num * frame.laser_num>>>(this->frame_.gpu()->points, channel_azimuths_cu_, channel_elevations_cu_, 
  point_data_cu_, sensor_timestamp_cu_, frame.distance_unit, this->transform_, this->optical_center, frame.block_num, frame.laser_num, frame.packet_num, frame.config.fov_start, frame.config.fov_end);
  cudaSafeCall(hipGetLastError(), ReturnCode::CudaXYZComputingError);
  this->frame_.DeviceToHost(0, frame.block_num * frame.laser_num * frame.packet_num * sizeof(T_Point));
  std::memcpy(frame.points, this->frame_.cpu()->points, frame.block_num * frame.laser_num * frame.packet_num * sizeof(T_Point));
  return 0;
}
template <typename T_Point>
int Udp1_4ParserGpu<T_Point>::LoadCorrectionString(char *correction_content) {
  if (corrections_loaded_) {
    return 0;
    if (channel_elevations_cu_) hipFree(channel_elevations_cu_);
    if (channel_azimuths_cu_) hipFree(channel_azimuths_cu_);
    corrections_loaded_ = false;
  }
  std::string correction_content_str = correction_content;
  std::istringstream ifs(correction_content_str);
  std::string line;
  // skip first line "Laser id,Elevation,Azimuth" or "eeff"
  std::getline(ifs, line);  

  float elevation_list[MAX_LASER_NUM], azimuth_list[MAX_LASER_NUM];

  std::vector<std::string> vfirstLine;
  split_string(vfirstLine, line, ',');
  if (vfirstLine[0] == "EEFF" || vfirstLine[0] == "eeff") {
    // skip second line
    std::getline(ifs, line);  
  }

  int lineCount = 0;
  while (std::getline(ifs, line)) {
    std::vector<std::string> vLineSplit;
    split_string(vLineSplit, line, ',');
    // skip error line or hash value line
    if (vLineSplit.size() < 3) {  
      continue;
    } else {
      lineCount++;
    }
    float elevation, azimuth;
    int laserId = 0;

    std::stringstream ss(line);
    std::string subline;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> laserId;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> elevation;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> azimuth;
    if (laserId > MAX_LASER_NUM || laserId <= 0) {
      LogFatal("laser id is wrong in correction file. laser Id: %d, line: %d", laserId, lineCount);
      continue;
    }
    if (laserId != lineCount) {
      LogWarning("laser id is wrong in correction file. laser Id: %d, line: %d.  continue", laserId, lineCount);
      continue;
    }
    elevation_list[laserId - 1] = elevation;
    azimuth_list[laserId - 1] = azimuth;
  }
  CUDACheck(hipMalloc(&channel_azimuths_cu_, sizeof(float) * MAX_LASER_NUM));
  CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(float) * MAX_LASER_NUM));
  CUDACheck(hipMemcpy(channel_azimuths_cu_, azimuth_list, sizeof(float) * MAX_LASER_NUM, hipMemcpyHostToDevice));
  CUDACheck(hipMemcpy(channel_elevations_cu_, elevation_list, sizeof(float) * MAX_LASER_NUM, hipMemcpyHostToDevice));
  corrections_loaded_ = true;
  return 0;
}
template <typename T_Point>
int Udp1_4ParserGpu<T_Point>::LoadCorrectionFile(std::string lidar_correction_file) {
  LogInfo("load correction file from local correction.csv now!");
  std::ifstream fin(lidar_correction_file);
  if (fin.is_open()) {
    int length = 0;
    fin.seekg(0, std::ios::end);
    length = static_cast<int>(fin.tellg());
    fin.seekg(0, std::ios::beg);
    char *buffer = new char[length];
    fin.read(buffer, length);
    fin.close();
    int ret = LoadCorrectionString(buffer);
    delete[] buffer;
    if (ret != 0) {
      LogError("Parse local Correction file Error");
    } else {
      LogInfo("Parse local Correction file Success!!!");
      return 0;
    }
  } else {
    LogError("Open correction file failed");
    return -1;
  }
  return -1;
}
