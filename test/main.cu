#include "hip/hip_runtime.h"
#include "hesai_lidar_sdk_gpu.cuh"
#define PCL_NO_PRECOMPILE
#include <pcl/point_types.h>
#include <pcl/visualization/pcl_visualizer.h>
#include <pcl/io/pcd_io.h>

// #define SAVE_PCD_FILE
// #define ENABLE_VIEWER

// using namespace hesai::lidar;
struct PointXYZIT {
  //添加pcl里xyz
  PCL_ADD_POINT4D   
  float intensity;
  double timestamp;
  uint16_t ring;                   
  EIGEN_MAKE_ALIGNED_OPERATOR_NEW  
} EIGEN_ALIGN16;                   

POINT_CLOUD_REGISTER_POINT_STRUCT(
    PointXYZIT,
    (float, x, x)(float, y, y)(float, z, z)(float, intensity, intensity)(
        double, timestamp, timestamp)(uint16_t, ring, ring))


using namespace pcl::visualization;
std::shared_ptr<PCLVisualizer> pcl_viewer;
uint32_t last_frame_time;
uint32_t cur_frame_time;std::mutex mex_viewer;


//log info, display frame message
void lidarCallback(const LidarDecodedFrame<PointXYZIT>  &frame) {
  cur_frame_time = GetTickCount();  
  if (cur_frame_time - last_frame_time > kMaxTimeInterval) {
    printf("Time between last frame and cur frame is: %d\n", (cur_frame_time - last_frame_time));
  }
  last_frame_time = cur_frame_time;    
  printf("frame:%d points:%u packet:%d start time:%lf end time:%lf\n",frame.frame_index, frame.points_num, frame.packet_num, frame.points[0].timestamp, frame.points[frame.points_num - 1].timestamp) ;
  pcl::PointCloud<PointXYZIT>::Ptr pcl_pointcloud(new pcl::PointCloud<PointXYZIT>);
  mex_viewer.lock();
  if (frame.points_num == 0) return;
  pcl_pointcloud->clear();
  pcl_pointcloud->resize(frame.points_num);
  pcl_pointcloud->points.assign(frame.points, frame.points + frame.points_num);
  pcl_pointcloud->height = 1;
  pcl_pointcloud->width = frame.points_num;
  pcl_pointcloud->is_dense = false;
  pcl::PCDWriter writer;
  std::string file_name = "./PointCloudFrame" + std::to_string(frame.frame_index) + ".pcd";

//save point cloud with pcd file if define SAVE_PCD_FILE
#ifdef SAVE_PCD_FILE
  writer.writeASCII(file_name, *pcl_pointcloud);
#endif  

//display point cloud with pcl if define ENABLE_VIEWER
#ifdef ENABLE_VIEWER   
  PointCloudColorHandlerGenericField<PointXYZIT> point_color_handle(pcl_pointcloud, "intensity");
  pcl_viewer->updatePointCloud<PointXYZIT>(pcl_pointcloud, point_color_handle, "pandar");
#endif
mex_viewer.unlock();
}

//display point cloud with pcl if define ENABLE_VIEWER
void PclViewerInit(std::shared_ptr<PCLVisualizer> pcl_viewer) {
  pcl_viewer = std::make_shared<PCLVisualizer>("HesaiPointCloudViewer");
  pcl_viewer->setBackgroundColor(0.0, 0.0, 0.0);
  pcl_viewer->addCoordinateSystem(1.0);
  pcl::PointCloud<PointXYZIT>::Ptr pcl_pointcloud(new pcl::PointCloud<PointXYZIT>);
  pcl_viewer->addPointCloud<PointXYZIT>(pcl_pointcloud, "pandar");
  pcl_viewer->setPointCloudRenderingProperties(PCL_VISUALIZER_POINT_SIZE, 2, "pandar");
  return;
}

int main(int argc, char *argv[])
{
  #ifdef ENABLE_VIEWER   
  PclViewerInit(pcl_viewer);
  #endif 
  HesaiLidarSdkGpu<PointXYZIT> sample;
  DriverParam param;
  // assign param
  param.decoder_param.enable_parser_thread = false;
  param.input_param.source_type = DATA_FROM_LIDAR;
  param.input_param.pcap_path = "Your pcap file path";
  param.input_param.correction_file_path = "Your correction file path";
  param.input_param.firetimes_path = "Your firetime file path";

  param.input_param.device_ip_address = "192.168.1.201";
  param.input_param.ptc_port = 9347;
  param.input_param.udp_port = 2368;
  param.input_param.host_ip_address = "192.168.1.100";
  param.input_param.multicast_ip_address = "";

  //init lidar with param
  sample.Init(param);

  //assign callback fuction
  sample.RegRecvCallback(lidarCallback);

  //star process thread
  sample.Start();
  while (1)
  {

#ifdef ENABLE_VIEWER   
    mex_viewer.lock();
    if(pcl_viewer->wasStopped()) break;
    pcl_viewer->spinOnce();
    mex_viewer.unlock();
#endif     
    std::this_thread::sleep_for(std::chrono::milliseconds(40));

  }
}